#include <hip/hip_runtime.h>
#include <stdio.h>
#include <math.h>
#define SIZ 1024
__global__ //이게 device에서 실행될 function 각 thread가 일정량실행
void countnum(int* countarr,int* datarr,int n){ //threadIdx.x+blockDim.x*blockIdx.x
	int i=threadIdx.x+blockDim.x * blockIdx.x;
	if(i<n)
	{
		int num=datarr[i];
		atomicAdd(&countarr[num],1);
	}

	//atomicAdd(&arr[0],1);
	//printf(" %d ",arr[0]);

}
/*__global__
void countbignum(int* countarr,int* datarr,int n){

	int offset=1;
	if(offset<n)
	{
		int num=datarr[offset];
		atomicAdd(&countarr[num],1);
	}
}*/
__host__ void counting_sort(int arr[], int size, int max_val)
{
   // fill in 
	int* counting;
	counting=(int*)malloc(sizeof(int)*size);
	int* counting_d;
	int* datarr_d;//input array
	hipMalloc((void **)&counting_d,sizeof(int)*max_val);
	hipMemset(counting_d,0,max_val*(sizeof(int)));
	hipMalloc((void**) &datarr_d,sizeof(int)*size);
	hipMemcpy(datarr_d,arr,sizeof(int)*size,hipMemcpyHostToDevice);
	int blocknum;
	blocknum=ceil((double)size/SIZ);

	//	countnum<<<1024,SIZ>>>(counting_d,datarr_d,size);
	//countnum<<<65535,SIZ>>>(counting_d,datarr_d,size);//1024가 max
	//	countnum<<<2097152,SIZ>>>(counting_d,datarr_d,size);
	//countnum<<<1048576,SIZ>>>(counting_d,datarr_d,size);
	countnum<<<blocknum,SIZ>>>(counting_d,datarr_d,size);
	
	hipDeviceSynchronize();
	hipMemcpy(counting,counting_d,sizeof(int)*max_val,hipMemcpyDeviceToHost);
	int index;
	index=0;
	for(int j=0;j<max_val;j++)
	{
		for(int q=0;q<counting[j];q++)
		{
			arr[index++]=j;
		}
	}
	free(counting);
	hipFree(counting_d);
	hipFree(datarr_d);

}

